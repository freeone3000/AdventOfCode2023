//
// Created by jasmine on 12/31/23.
// This obviously is implementing ray tracing. So, let's see about RTX here.
//

#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

int main() {
    return 0;
}