#include "hip/hip_runtime.h"
//
// Created by jasmine on 12/31/23.
// This obviously is implementing ray tracing. So, let's see about RTX here.
//

#pragma clang diagnostic push
#pragma ide diagnostic ignored "OCUnusedMacroInspection"
#pragma ide diagnostic ignored "bugprone-reserved-identifier"
// for sscanf_s
#define __STDC_WANT_LIB_EXT1__ 1
#include <cstdio>
#pragma clang diagnostic pop

#include <unistd.h>
#include <iostream>
#include <utility>
#include <vector>
#include <string>

// cuda
#include <hip/hip_runtime.h>

#include "lines.hpp"

using std::nullopt;

#define CHECK_CUDA(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// max size for position real problem is 400000000000000, which is 2**48.5. We need 64 bit.
// but we can fit velocity in a short!
// for alignas, see https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#vector-types-alignment-requirements-in-device-code
struct alignas(8) Hailstone {
public:
    long3 position;
    short3 velocity;
    /** for printing */
    friend std::ostream& operator<<(std::ostream& os, const Hailstone& hailstone) {
        os << hailstone.position.x << ", " << hailstone.position.y << ", " << hailstone.position.z << " @ " <<
           hailstone.velocity.x << ", " << hailstone.velocity.y << ", " << hailstone.velocity.z;
        return os;
    }
};

float time_intersect(long p0, long p1, short v0, short v1) {
    // p0 + v0 * t = p1 + v1 * t
    // p0 - p1 = (v1 - v0) * t
    // (p0 - p1) / (v1 - v0) = t
    // if v0 == v1, then we have a collision
    // if v0 != v1, then we have a collision at t = (p0 - p1) / (v1 - v0)
    // if v0 == v1, then we have a collision at t = 0
    if(v0 == v1) {
        return 0;
    }
    return static_cast<float>(p0 - p1) / static_cast<float>(v1 - v0);
}

float position_at_time(long p, short v, float t) {
    return static_cast<float>(p) + static_cast<float>(v) * t;
}

std::optional<float2> check_intersection(Hailstone a, Hailstone b, const long2& disc) {
    //check x
    float t = time_intersect(a.position.x, b.position.x, a.velocity.x, b.velocity.x);
    if(t < 0) {
        return nullopt;
    }
    float x0 = position_at_time(a.position.x, a.velocity.x, t);
    float x1 = position_at_time(b.position.x, b.velocity.x, t);
    if(x0 != x1 || static_cast<long>(x0) < disc.x || static_cast<long>(x0) > disc.y) {
        return nullopt;
    }
    //check y
    t = time_intersect(a.position.y, b.position.y, a.velocity.y, b.velocity.y);
    if(t < 0) {
        return nullopt;
    }
    float y0 = position_at_time(a.position.y, a.velocity.y, t);
    float y1 = position_at_time(b.position.y, b.velocity.y, t);
    if(y0 != y1 || static_cast<long>(y0) < disc.x || static_cast<long>(y0) > disc.y) {
        return nullopt;
    }
    return make_float2(x0, y0);
}

#pragma clang diagnostic push
#pragma ide diagnostic ignored "cert-err34-c" // ignore sscanf
int raytrace_on_file(std::vector<std::string> items, long2 target_minmax) {
    const size_t num_hailstones = items.size();
    Hailstone hailstones[num_hailstones];

    size_t hail_idx = 0;
    for(auto it = items.begin(); it != items.end(); ++it, ++hail_idx) {
        int64_t x, y, z;
        int16_t vx, vy, vz;
#ifdef __STDC_LIB_EXT1__
        sscanf_s((*it).c_str(), "%ld, %ld, %ld @ %hd, %hd, %hd", &x, &y, &z, &vx, &vy, &vz, item.length());
#else
        sscanf((*it).c_str(), "%ld, %ld, %ld @ %hd, %hd, %hd", &x, &y, &z, &vx, &vy, &vz);
#endif //__STDC_LIB_EXT1__
        hailstones[hail_idx] = Hailstone{make_long3(x, y, z), make_short3(vx, vy, vz)};
    }


    int count = 0;
    for(size_t i = 0; i < num_hailstones; ++i) {
        for(size_t j = i + 1; j < num_hailstones; ++j) {
            auto intersection = check_intersection(hailstones[i], hailstones[j], target_minmax);
            if(intersection) {
                std::cout << "Hailstone A: " << hailstones[i] << std::endl;
                std::cout << "Hailstone B: " << hailstones[j] << std::endl;
                std::cout << "Intersect at (" << intersection->x << ", " << intersection->y << ")\n" << std::endl;
                ++count;
            }
        }
    }
    return count;
}
#pragma clang diagnostic pop

int main(int argc, const char* argv[]) {
    if(argc != 4) {
        std::cerr << "Usage: " << argv[0] << " [file] [target-min] [target-max]" << std::endl;
        return 1;
    }
    auto target = make_long2(atol(argv[2]), atol(argv[3]));

    int res = run_function_on_file<int>(argv[1], [target](std::vector<std::string> vec){
        return raytrace_on_file(std::move(vec), target);
    });
    std::cout << "Result: " << res << std::endl;
}